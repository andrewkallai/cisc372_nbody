#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"

__global__
void matrix_accels(vector3* d_accels, double* d_hPos, double* d_mass) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < NUMENTITIES && j < NUMENTITIES) {
    if (i == j) {
        FILL_VECTOR(d_accels[i*NUMENTITIES + j], 0, 0, 0);
    } else {
        vector3 distance;
        for (int k = 0; k < 3; ++k) distance[k] = d_hPos[i*3 + k] - d_hPos[j*3 + k];
        double magnitude_sq = distance[0]*distance[0] + distance[1]*distance[1] + distance[2]*distance[2];
        double magnitude = sqrt(magnitude_sq);
        double accelmag = -1 * GRAV_CONSTANT * d_mass[j] / magnitude_sq;
        FILL_VECTOR(d_accels[i*NUMENTITIES + j], accelmag * distance[0] / magnitude, accelmag * distance[1] / magnitude, accelmag * distance[2] / magnitude);
    }
  }
}

__global__
void matrix_sums(vector3* d_accels, double* d_hPos, double* d_hVel, double* d_mass) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < NUMENTITIES) {
    vector3 accel_sum = {0, 0, 0};
    for (int j = 0; j < NUMENTITIES; ++j) {
      for (int k = 0; k < 3; ++k)
        accel_sum[k] += d_accels[i*NUMENTITIES + j][k];
    }

    for (int k = 0; k < 3; ++k) {
      d_hVel[i*3 + k] += accel_sum[k] * INTERVAL;
      d_hPos[i*3 + k] += d_hVel[i*3 + k] * INTERVAL;
    }
  }
}
//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){
	//make an acceleration matrix which is NUMENTITIES squared in size;
/*
	int i,j,k;
	vector3* values=(vector3*)malloc(sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	vector3** accels=(vector3**)malloc(sizeof(vector3*)*NUMENTITIES);
	for (i=0;i<NUMENTITIES;i++)
		accels[i]=&values[i*NUMENTITIES];
	//first compute the pairwise accelerations.  Effect is on the first argument.
	for (i=0;i<NUMENTITIES;i++){
		for (j=0;j<NUMENTITIES;j++){
			if (i==j) {
				FILL_VECTOR(accels[i][j],0,0,0);
			}
			else{
				vector3 distance;
				for (k=0;k<3;k++) distance[k]=hPos[i][k]-hPos[j][k];
				double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
				double magnitude=sqrt(magnitude_sq);
				double accelmag=-1*GRAV_CONSTANT*mass[j]/magnitude_sq;
				FILL_VECTOR(accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
			}
		}
	}
	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	for (i=0;i<NUMENTITIES;i++){
		vector3 accel_sum={0,0,0};
		for (j=0;j<NUMENTITIES;j++){
			for (k=0;k<3;k++)
				accel_sum[k]+=accels[i][j][k];
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (k=0;k<3;k++){
			hVel[i][k]+=accel_sum[k]*INTERVAL;
			hPos[i][k]+=hVel[i][k]*INTERVAL;
		}
	}
	free(accels);
	free(values);
*/
    vector3 *d_accels;
    double *d_hPos, *d_hVel, *d_mass;

    hipMalloc((void **)&d_accels, sizeof(vector3) * NUMENTITIES * NUMENTITIES);
    hipMalloc((void **)&d_hPos, sizeof(double) * NUMENTITIES * 3);
    hipMalloc((void **)&d_hVel, sizeof(double) * NUMENTITIES * 3);
    hipMalloc((void **)&d_mass, sizeof(double) * NUMENTITIES);

    hipMemcpy(d_hPos, hPos, sizeof(double) * NUMENTITIES * 3, hipMemcpyHostToDevice);
    hipMemcpy(d_hVel, hVel, sizeof(double) * NUMENTITIES * 3, hipMemcpyHostToDevice);
    hipMemcpy(d_mass, mass, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((NUMENTITIES + threadsPerBlock.x - 1) / threadsPerBlock.x, (NUMENTITIES + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_accels<<<blocksPerGrid, threadsPerBlock>>>(d_accels, d_hPos, d_mass);

    matrix_sums<<<(NUMENTITIES + threadsPerBlock.x - 1) / threadsPerBlock.x, threadsPerBlock.x>>>(d_accels, d_hPos, d_hVel, d_mass);
    hipDeviceSynchronize();

    hipMemcpy(hPos, d_hPos, sizeof(double) * NUMENTITIES * 3, hipMemcpyDeviceToHost);
    hipMemcpy(hVel, d_hVel, sizeof(double) * NUMENTITIES * 3, hipMemcpyDeviceToHost);

    hipFree(d_accels);
    hipFree(d_hPos);
    hipFree(d_hVel);
    hipFree(d_mass);
}
